//To compile should add "-lfftw3 -lgsl"

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <math.h>
// #include <gsl/gsl_rng.h>
#include <cstdio>
#include <cstdlib>
#include <string>
#include <iostream>
#include <time.h>
//#include <fftw3.h>
#include <complex>
#include <hipfft/hipfft.h>
using namespace std;

struct particle3D{
	int number;
	double *mass;
	double *x;
	double *y;
	double *z;
	double *Fx;
	double *Fy;
	double *Fz;
	double *vx;
	double *vy;
	double *vz;
};
struct grid3D{
	double L;
	int Nx;
	int Ny;
	int Nz;
	int N;
	double dx;
	double dy;
	double dz;
	double *phi;
	double *density;
	double *Fx;
	double *Fy;
	double *Fz;
};
struct rk43D{
    int step1;
    int step2;
    double *ax;
    double *ay;
	double *az;
    double *vx;
    double *vy;
	double *vz;
};
__global__
void phiToForce(double* phi,double* Fx,double* Fy,double* Fz,int Nx,double L,double nConst){
	int N = Nx*Nx*Nx;
	int const Ny = Nx;
  	int const Nz = Nx;
	double dx = L / Nx;
	double factor = -1./(2.0*dx) * nConst;

	int index = blockDim.x * blockIdx.x + threadIdx.x;
    
    while(index < N){
    	int ii = index / (Ny*Nz);
    	int jj = (index / Nz) % Ny;
    	int kk = index % Nz;
    	Fx[index] = factor*( phi[ ( (ii+1)%Nx )*Ny*Nz + jj*Nz + kk ] 
                              - phi[ ( (Nx+ii-1)%Nx )*Ny*Nz + jj*Nz + kk ] );

    	Fy[index] = factor*( phi[ ii*Ny*Nz + ( (jj+1)%Ny )*Nz + kk ] 
                              - phi[ ii*Ny*Nz + ( (Ny+jj-1)%Ny )*Nz + kk ] );  

  		Fz[index] = factor*( phi[ ii*Ny*Nz  + jj*Nz + ((kk+1)%Nz)] 
                              - phi[ ii*Ny*Nz  + jj*Nz + ((Nz+kk-1)%Nz)]);
    	index += blockDim.x*gridDim.x;
    }

}
__global__
void overk2(double *out,int Nx,int Ny,int Nzh){
	//Notice that *out actullay is a compelx <double> pointer array.

	int N = Nx*Ny*Nzh;
	int index = blockDim.x * blockIdx.x + threadIdx.x;
	int fi,fj;
	double kxx,kyy,kzz;

    while(index < N){
    	int ii = index / (Ny*Nzh);
    	int jj = (index / Nzh) % Ny;
    	int kk = index % Nzh;

    	if (2*ii < Nx) {fi = ii;}
		else           {fi = Nx-ii;}
		if (2*jj < Ny) {fj = jj;}
		else           {fj = Ny-jj;}

		kxx = 1.0*fi*fi;
		kyy = 1.0*fj*fj;
		kzz = 1.0*kk*kk;

		if(index != 0){
			//ii != 0 || jj != 0 || kk!=0
			out[2*index] = out[2*index] / (kxx+kyy+kzz);		//real part
			out[2*index+1] = out[2*index+1] / (kxx+kyy+kzz);	//imaginary part
		}   	

    	index += blockDim.x*gridDim.x;
    }
}

void Weight(struct grid3D *grid,struct particle3D *particle,int type);
void WeightForce(struct grid3D *grid,struct particle3D *particle,int type);
void poisson_solver_fft_force_3d(int const dim, struct grid3D *grid);
void _2nd_order_diff_3d(struct grid3D *grid, int const ii, int const jj, int const kk );
void _2nd_order_diff_3d_cuda(struct grid3D *grid);
void calculateGreenFFT(struct grid3D *grid, complex<double>* fftgf);
void isolatedPotential(struct grid3D *grid, complex<double>* fftgf);


void kick(struct particle3D *particle , double dt);
void drift(struct particle3D *particle , double dt);
void init_rk4(struct particle3D *particle, struct particle3D *buff, struct rk43D *rk4);
void rk4_mid(struct particle3D *particle, struct particle3D *buff, struct rk43D *rk4, double dt, int weighting);
void rk4_end(struct particle3D *particle, struct rk43D *rk4, double dt);
void periodic_boundary(double position, double length);
void boundary_check(int boundary, struct particle3D *particle, double L);

//Functions to locate memory and free memory of different struct.
void locateMemoryParticle(struct particle3D *particle,int N);
void freeMemoryParticle(struct particle3D *particle);
void locateMemoryRk4(struct rk43D *rk4,int N);
void freeMemoryRk4(struct rk43D *rk4);
void locateMemoryGrid(struct grid3D *grid);
void freeMemoryGrid(struct grid3D *grid);


int main( int argc, char *argv[] ){
	//================Simulation Constants
	int weightFunction = 1;  	//0/1/2 : NGP/CIC/TSC
	int orbitIntegration = 1;	//0/1/2 : KDK/DKD/RK4
	int poissonSolver = 0;		//0/1   : fft/isolated
	int boundary = 0;           //0/1/2 : periodic/isolated/no boundary
	int dim = 3;				
	double L = 10.0;				//Length of box (from -L/2 ~ L/2)
	int Nx = 256;				//Number of grid in x direction. (should be odd number)
	int NParticle=2;//Number of particles used in simulation
	//double massParticle=1.0;
	double dt = 1.0e-2;
	double G = 1.0;
	double T,r1_0,r2_0;
	hipEvent_t start, stop;	//For cuda timing
	float totalTime;
	
	//================Structs
	struct grid3D grid;
	struct particle3D myParticle;
	struct particle3D buffParticle;		//If not RK4 mode , it will not be malloc and free
	struct rk43D myrk4;					//If not RK4 mode , it will not be malloc and free
	complex<double>* fftgf;

	hipEventCreate(&start);
	hipEventCreate(&stop);
 	hipEventRecord(start,0);

 	

	//Output to a file
	FILE *output;
	output = fopen("result.txt","w");
	//================Random number generator.
		//To use : d=gsl_rng_uniform(rng);
		// gsl_rng *rng;
		// rng = gsl_rng_alloc(gsl_rng_mt19937);
		// gsl_rng_set(rng,123456);//The seed is 123456.

	//================Initialize Grid Parameter=========== 
		
		grid.L = L;		
		grid.Nx = Nx;		
		grid.Ny = grid.Nx;
		grid.Nz = grid.Nx;
		grid.N = grid.Nx * grid.Ny * grid.Nz;
		grid.dx = grid.L / (grid.Nx-1);	//-1 is because boundary (make box closed)
		grid.dy = grid.L / (grid.Ny-1);
		grid.dz = grid.L / (grid.Nz-1);
		locateMemoryGrid(&grid);

	//================Initialize Particles ======
		
		myParticle.number = NParticle;
		locateMemoryParticle(&myParticle,NParticle);

		if(orbitIntegration == 2){
	//================Initialize Particles for RK4 (buffer)======
			buffParticle.number = NParticle;
			locateMemoryParticle(&buffParticle,NParticle);
	//================Initialize Runge-Kutta Coefficient======
			myrk4.step1 = 0;
			myrk4.step2 = 1;
			locateMemoryRk4(&myrk4,NParticle);
		}

		if(poissonSolver == 1){
			fftgf = (complex<double>*) malloc(sizeof(complex<double>) * 8*grid.N);
			calculateGreenFFT(&grid,fftgf);
		}
      
	
	//Initialize mass of particles
		myParticle.mass[0]=2.0;
		myParticle.mass[1]=1.0;

		if(orbitIntegration == 2){
			buffParticle.mass[0] = myParticle.mass[0];
			buffParticle.mass[1] = myParticle.mass[1];
		}
	//Initialize Initial Position of Particle.
		// for (int i = 0; i < myParticle.number; ++i){
		// 	myParticle.x[i]=gsl_rng_uniform(rng) * grid.L - grid.L/2;
		// 	myParticle.y[i]=gsl_rng_uniform(rng) * grid.L - grid.L/2;
		// 	printf("At (%f,%f) \n",myParticle.x[i],myParticle.y[i]);
		// }
		myParticle.x[0] = 1.0;
		myParticle.y[0] = 0.0;
		myParticle.z[0] = 0.0;
		myParticle.x[1] = -2.0;
		myParticle.y[1] = 0.0;
		myParticle.z[1] = 0.0;
		r1_0 = sqrt(pow(myParticle.x[0],2)+pow(myParticle.y[0],2)+pow(myParticle.z[0],2));
		r2_0 = sqrt(pow(myParticle.x[1],2)+pow(myParticle.y[1],2)+pow(myParticle.z[1],2));

		Weight(&grid,&myParticle,weightFunction);
		
		if(poissonSolver == 0){poisson_solver_fft_force_3d(dim,&grid);}
		else if ( poissonSolver == 1 ){isolatedPotential(&grid,fftgf);}
		
		WeightForce(&grid,&myParticle,weightFunction);

	// //Initialize Initial velocity
		myParticle.vx[0] = 0.0;
		myParticle.vy[0] = -sqrt(fabs(myParticle.Fx[0]*myParticle.x[0])/myParticle.mass[0]);
		myParticle.vz[0] = 0.0;
		myParticle.vx[1] = 0.0;
		myParticle.vy[1] = sqrt(fabs(myParticle.Fx[1]*myParticle.x[1])/myParticle.mass[1]);
		myParticle.vz[1] = 0.0;

		double F_0;
		F_0 = G * myParticle.mass[0] * myParticle.mass[1]/pow(r1_0+r2_0,2);
		T =sqrt(myParticle.mass[0]*4*pow(M_PI,2)*r1_0/F_0);

		//Check whether force is same magnitude but inverse direction.
		printf("%f\t%f\t%f\n",myParticle.Fx[0],myParticle.Fy[0],myParticle.Fz[0]);
		printf("%f\t%f\t%f\n",myParticle.Fx[1],myParticle.Fy[1],myParticle.Fz[1]);
		
	//Time evolution loop
	double t = 0.0;
	for(int st=0;st < 1000;st++){
	 	//Deposit Particles to grid
	 	Weight(&grid,&myParticle,weightFunction);

	 	//Use Fourier Transform to calculate potential and force.
		if ( poissonSolver == 0 )	poisson_solver_fft_force_3d(dim, &grid);
		else if ( poissonSolver == 1 ){isolatedPotential(&grid,fftgf);}

 		//Remap the force to particle.
		WeightForce(&grid,&myParticle,weightFunction);

      

 		//Move particle
		if(orbitIntegration == 0){
 		//KDK scheme
			kick(&myParticle,dt/2);
 			drift(&myParticle,dt);
			boundary_check(boundary, &myParticle, L);
 			Weight(&grid,&myParticle,weightFunction);
			if ( poissonSolver == 0 )       poisson_solver_fft_force_3d(dim, &grid);
			else if ( poissonSolver == 1 ){isolatedPotential(&grid,fftgf);}
			WeightForce(&grid,&myParticle,weightFunction);

 			kick(&myParticle,dt/2);
		}
		else if(orbitIntegration == 1){
 			//DKD scheme
 			drift(&myParticle,dt/2);
			boundary_check(boundary, &myParticle, L);
 			Weight(&grid,&myParticle,weightFunction);
			if ( poissonSolver == 0 )       poisson_solver_fft_force_3d(dim, &grid);
			else if ( poissonSolver == 1 )  isolatedPotential(&grid,fftgf);
			WeightForce(&grid,&myParticle,weightFunction);

 			kick(&myParticle,dt);
 			drift(&myParticle,dt/2);
 		}
		else if(orbitIntegration == 2){
 			//RK4	
			init_rk4(&myParticle,&buffParticle,&myrk4);
			
			rk4_mid(&myParticle,&buffParticle,&myrk4,dt/2,1);        //k1
			boundary_check(boundary, &buffParticle, L);
			Weight(&grid,&buffParticle,weightFunction);
			if ( poissonSolver == 0 )       poisson_solver_fft_force_3d(dim, &grid);
			else if ( poissonSolver == 1 )  isolatedPotential(&grid,fftgf);
			WeightForce(&grid,&buffParticle,weightFunction);
			
			rk4_mid(&myParticle,&buffParticle,&myrk4,dt/2,2);        //k2
			boundary_check(boundary, &buffParticle, L);
			Weight(&grid,&buffParticle,weightFunction);
			if ( poissonSolver == 0 )       poisson_solver_fft_force_3d(dim, &grid);
			else if ( poissonSolver == 1 )  isolatedPotential(&grid,fftgf);
			WeightForce(&grid,&buffParticle,weightFunction);
			
			rk4_mid(&myParticle,&buffParticle,&myrk4,dt,2);          //k3
			boundary_check(boundary, &buffParticle, L);
			Weight(&grid,&buffParticle,weightFunction);
			if ( poissonSolver == 0 )       poisson_solver_fft_force_3d(dim, &grid);
			else if ( poissonSolver == 1 )  isolatedPotential(&grid,fftgf);
			WeightForce(&grid,&buffParticle,weightFunction);
			
			rk4_mid(&myParticle,&buffParticle,&myrk4,dt,1);          //k4
			
			rk4_end(&myParticle,&myrk4,dt);

 		}
		//Boundary Condition
		#ifdef CHECK_THIS_LATER
		if (boundary == 0){
			for (int i=0; i<NParticle; i++){
			       	if ( abs(myParticle.x[i]) > L/2){
					periodic_boundary(myParticle.x[i],L);
				}
				if ( abs(myParticle.y[i]) > L/2){
					periodic_boundary(myParticle.y[i],L);
				}
				if ( abs(myParticle.z[i]) > L/2){
					periodic_boundary(myParticle.z[i],L);
				}
			}	
		}
		else if (boundary == 1){
			for (int i=0; i<NParticle; i++){
				if ( abs(myParticle.x[i]) > L/2 || abs(myParticle.y[i]) > L/2 || abs(myParticle.z[i]) > L/2){
					myParticle.mass[i] = 0;
					cout << "A particle reaches the boundary." << endl;
				}
			}
		}
 		#endif
		boundary_check(boundary, &myParticle, L);

 		//print out the position of particle 1
		if(st % 20 == 0){
			printf("Step:%d\n", st);
			// double momentum_x = 0;
			// double momentum_y = 0;
			// double momentum_z = 0;

   //      		for (int i=0; i<NParticle; i++){
			// 	momentum_x += myParticle.mass[i] * myParticle.vx[i];
			// 	momentum_y += myParticle.mass[i] * myParticle.vy[i];
			// 	momentum_z += myParticle.mass[i] * myParticle.vz[i];
			// }
			// cout << "(px , py,  pz) = (" << momentum_x << ", " << momentum_y << ", " << momentum_z << ")" << endl;
			fprintf(output,"%f\t%f\t",myParticle.x[0],myParticle.y[0]);
			fprintf(output,"%f\t%f\n",myParticle.x[1],myParticle.y[1]);
	
		}
		t+=dt;
 	}
 	hipEventRecord(stop,0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&totalTime, start, stop);
	printf("Total time : %f\n",totalTime/1000);
		
 	fclose(output);
	freeMemoryGrid(&grid);
	freeMemoryParticle(&myParticle);
	if(orbitIntegration == 2){
		freeMemoryParticle(&buffParticle);
	}
	if(poissonSolver == 1){
		free(fftgf);
	}
	return 0;
}

void poisson_solver_fft_force_3d(int const dim, struct grid3D *grid){
	
	int const Nx = grid->Nx;
	int const Ny = grid->Ny; 
	int const Nz = grid->Nz;
	int const Nzh = (Nz/2+1);
	hipfftHandle p1,p2;
	
	double *in;
	complex<double> *out;
	in = (double*) malloc( sizeof(double) * Nx*Ny*Nz );
	out = (complex<double>*) malloc( sizeof(complex<double>) * Nx*Ny*Nzh);
		

	

	/////////// fft ///////////
	hipfftDoubleReal *dataIn;
	hipfftDoubleComplex *dataOut;
    hipMalloc((void**)&dataIn, sizeof(hipfftDoubleReal)*Nx*Ny*Nz);
    hipMalloc((void**)&dataOut, sizeof(hipfftDoubleComplex)*Nx*Ny*Nzh);
    hipMemcpy(dataIn, grid->density, sizeof(hipfftDoubleReal)*Nx*Ny*Nz, hipMemcpyHostToDevice);
	
	//cufft
	if (hipfftPlan3d(&p1,Nx,Ny,Nz, HIPFFT_D2Z) != HIPFFT_SUCCESS) {
        printf("CUFFT error: Plan D2Z creation failed.\n");
		exit(1);
    }
    if (hipfftExecD2Z(p1, dataIn, dataOut) != HIPFFT_SUCCESS) {
		printf("CUFFT error: ExecD2Z forward failed.\n");
		exit(1);
    }
    hipMemcpy(out, dataOut,sizeof(hipfftDoubleComplex)*Nx*Ny*Nzh, hipMemcpyDeviceToHost);
    hipDeviceSynchronize();
    hipfftDestroy(p1);
    

	double *d_out;
	hipMalloc((void**)&d_out, sizeof(double)*2*Nx*Ny*Nzh);
	hipMemcpy(d_out,out,sizeof(double)*2*Nx*Ny*Nzh,hipMemcpyHostToDevice);

	overk2 <<<128,128>>> (d_out,Nx,Ny,Nzh);

	hipMemcpy(out,d_out,sizeof(double)*2*Nx*Ny*Nzh,hipMemcpyDeviceToHost);

	hipFree(d_out);
	


	/////////// inverse fft ///////////
    hipMemcpy(dataOut, out,sizeof(hipfftDoubleComplex)*Nx*Ny*Nzh, hipMemcpyHostToDevice);
	
	//cufft
	if (hipfftPlan3d(&p2,Nx,Ny,Nz, HIPFFT_Z2D) != HIPFFT_SUCCESS) {
        printf("CUFFT error: Plan creation failed.\n");
		exit(1);
    }
    if (hipfftExecZ2D(p2, dataOut, dataIn) != HIPFFT_SUCCESS) {
		printf("CUFFT error: ExecZ2D  failed.\n");
		exit(1);
    }
    hipMemcpy(in, dataIn, sizeof(hipfftDoubleReal)*Nx*Ny*Nz, hipMemcpyDeviceToHost);
    hipDeviceSynchronize();
    hipfftDestroy(p2);


	/////////// normalization ///////////

	double nConst = -1.0 / M_PI/grid->L;	//Normalization constant.

	double* d_in;
	double* d_Fx; 
	double* d_Fy;
	double* d_Fz;

	int size = grid->N*sizeof(double);

	hipMalloc((void**)&d_in, size);
	hipMalloc((void**)&d_Fx, size);
	hipMalloc((void**)&d_Fy, size);
	hipMalloc((void**)&d_Fz, size);

	hipMemcpy(d_in,in,size,hipMemcpyHostToDevice);
	phiToForce <<<128,128>>>(d_in,d_Fx,d_Fy,d_Fz,grid->Nx,grid->L,nConst);
	hipDeviceSynchronize();
	hipMemcpy(grid->Fx,d_Fx,size, hipMemcpyDeviceToHost);
	hipMemcpy(grid->Fy,d_Fy,size, hipMemcpyDeviceToHost);
	hipMemcpy(grid->Fz,d_Fz,size, hipMemcpyDeviceToHost);

	hipFree(d_in);
	hipFree(d_Fx);
	hipFree(d_Fy);
	hipFree(d_Fz);
	hipFree(dataIn);
	hipFree(dataOut);
	

	free(in);
	free(out);
	
}


void _2nd_order_diff_3d(struct grid3D *grid, int const ii, int const jj, int const kk ) {

  double factor1 = -1./(2.*grid->dx);
  double factor2 = -1./(2.*grid->dy);
  double factor3 = -1./(2.*grid->dz);
  int const Nx = grid->Nx;
  int const Ny = grid->Ny;
  int const Nz = grid->Nz;
  int index = ii*Ny*Nz + jj*Nz + kk;

  grid->Fx[ index ] = factor1*( grid->phi[ ( (Nx+ii+1)%Nx )*Nx*Ny + jj*Nx + kk ] 
                              - grid->phi[ ( (Nx+ii-1)%Nx )*Nx*Ny + jj*Nx + kk ] );

  grid->Fy[ index ] = factor2*( grid->phi[ ii*Nx*Ny + ( (Ny+jj+1)%Ny )*Nx + kk ] 
                              - grid->phi[ ii*Nx*Ny + ( (Ny+jj-1)%Ny )*Nx + kk ] );  

  grid->Fz[ index ] = factor3*( grid->phi[ ii*Nx*Ny  + jj*Nx + ((Nz+kk+1)%Nz)] 
                              - grid->phi[ ii*Nx*Ny  + jj*Nx + ((Nz+kk-1)%Nz)]);

}
void _2nd_order_diff_3d_cuda(struct grid3D *grid) {
	double* d_phi;
	double* d_Fx; 
	double* d_Fy;
	double* d_Fz;

	int size = grid->N * sizeof(double);

	hipMalloc((void**)&d_phi, size);
	hipMalloc((void**)&d_Fx, size);
	hipMalloc((void**)&d_Fy, size);
	hipMalloc((void**)&d_Fz, size);

	hipMemcpy(d_phi,grid->phi,size,hipMemcpyHostToDevice);
	phiToForce <<<128,128>>>(d_phi,d_Fx,d_Fy,d_Fz,grid->Nx,grid->L,1.0);
	hipDeviceSynchronize();
	hipMemcpy(grid->Fx,d_Fx,size, hipMemcpyDeviceToHost);
	hipMemcpy(grid->Fy,d_Fy,size, hipMemcpyDeviceToHost);
	hipMemcpy(grid->Fz,d_Fz,size, hipMemcpyDeviceToHost);

	hipFree(d_phi);
	hipFree(d_Fx);
	hipFree(d_Fy);
	hipFree(d_Fz);
}

void calculateGreenFFT(struct grid3D *grid, complex<double> *fftgf){
	double *greenFunction;
	int N = grid->N;
	int Nx = grid->Nx;
	int Ny = grid->Ny;
	int Nz = grid->Nz;

	int NNx = 2*Nx;
	int NNy = 2*Ny;
	int NNz = 2*Nz;

	greenFunction = (double*)malloc(8*N*sizeof(double));
	

	complex<double> *gf; 	//For green function
	gf = (complex<double>*) malloc(sizeof(complex<double>) * 8*N);
	

	//Initialize green function array
	for(int i=0;i<NNx;i++){
		for(int j=0;j<NNy;j++){
			for(int k=0;k<NNz;k++){
				greenFunction[i*NNx*NNy+j*NNx+k] = 0.0;	
			}		
		}
	}
	//Construct green function
	for(int i=0;i<Nx;i++){
		for(int j=0;j<Ny;j++){
			for(int k=0;k<Nz;k++){
				if(i != 0 || j != 0 || k != 0){
					greenFunction[i*NNx*NNy+j*NNx+k] = -1.0/sqrt(pow(i,2)+pow(j,2)+pow(k,2));
				}
			}
		}
	}

	for(int i=Nx+1;i<NNx;i++){
		for(int j=0;j<Ny;j++){
			for(int k=0;k<Nz;k++){
				greenFunction[i*NNx*NNy+j*NNx+k] = greenFunction[(NNx-i)*NNx*NNy+j*NNx+k];
			}
		}
	}
	for(int i=0;i<Nx;i++){
		for(int j=Ny+1;j<NNy;j++){
			for(int k=0;k<Nz;k++){
				greenFunction[i*NNx*NNy+j*NNx+k] = greenFunction[i*NNx*NNy+(NNy-j)*NNx+k];
			}
		}
	}

	for(int i=Nx+1;i<NNx;i++){
		for(int j=Ny+1;j<NNy;j++){
			for(int k=0;k<Nz;k++){
				greenFunction[i*NNx*NNy+j*NNx+k] = greenFunction[(NNx-i)*NNx*NNy+(NNy-j)*NNx+k];
			}
			
		}
	}

	for(int i=0;i<NNx;i++){
		for(int j=0;j<NNy;j++){
			for(int k=Nz+1;k<NNz;k++){
				greenFunction[i*NNx*NNy+j*NNx+k] = greenFunction[i*NNx*NNy + j*NNx + (NNz-k)];
			}
		}
	}

	for(int i=0;i<NNx;i++){
		for(int j=0;j<NNy;j++){
			for(int k=0;k<NNz;k++){
				gf[i*NNx*NNy + j*NNx + k] = complex<double>(0.,0.);
				gf[i*NNx*NNy + j*NNx + k] += greenFunction[i*NNx*NNy + j*NNx + k];
			}
			
		}
	}

	hipfftHandle plan;
	hipfftDoubleComplex *data;
    hipMalloc((void**)&data, sizeof(hipfftDoubleComplex)*8*N);
    hipMemcpy(data, gf, sizeof(double)*8*N*2, hipMemcpyHostToDevice);

	if (hipfftPlan3d(&plan,NNx,NNy,NNz, HIPFFT_Z2Z) != HIPFFT_SUCCESS) {
        printf("CUFFT error: Plan creation failed.\n");
		exit(1);
    }
    if (hipfftExecZ2Z(plan, data, data, HIPFFT_FORWARD) != HIPFFT_SUCCESS) {
		printf("CUFFT error: ExecZ2Z forward failed.\n");
		exit(1);
    }
    hipMemcpy(fftgf, data, sizeof(double)*8*N*2, hipMemcpyDeviceToHost);
    hipDeviceSynchronize();
    hipfftDestroy(plan);
    hipFree(data);
	
	free(gf);
	free(greenFunction);
	
}

void isolatedPotential(struct grid3D *grid, complex<double>* fftgf){
	double *densityPad;
	int N = grid->N;
	int Nx = grid->Nx;
	int Ny = grid->Ny;
	int Nz = grid->Nz;

	int NNx = 2*Nx;
	int NNy = 2*Ny;
	int NNz = 2*Nz;

	densityPad = (double*)malloc(8*N*sizeof(double));
	hipfftHandle p1,p2;				//fft plan for cuFFT
	
	complex<double> *dp, *fftdp;	//For Padding density
	
	complex<double> *phi,*ifftphi;	//For potential
	
	dp = (complex<double>*) malloc(sizeof(complex<double>) * 8*N);
	fftdp = (complex<double>*) malloc(sizeof(complex<double>) * 8*N);
	phi = (complex<double>*) malloc(sizeof(complex<double>) * 8*N);
	ifftphi = (complex<double>*) malloc(sizeof(complex<double>) * 8*N);

	for(int i=0;i<NNx;i++){
		for(int j=0;j<NNy;j++){
			for(int k=0;k<NNz;k++){
				if(i < Nx && j < Ny && k < Nz){
					//Copy initial density
					densityPad[i*NNx*NNy+j*NNx+k] = grid->density[i*Nx*Ny+j*Nx+k];
				}else{
					//Padding 0s
					densityPad[i*NNx*NNy+j*NNx+k] = 0.0;
				}
			}
		}
	}
	

	for(int i=0;i<NNx;i++){
		for(int j=0;j<NNy;j++){
			for(int k=0;k<NNz;k++){
				dp[i*NNx*NNy + j*NNx + k] = complex<double>(0.,0.);
				dp[i*NNx*NNy + j*NNx + k] += densityPad[i*NNx*NNy + j*NNx + k];
			}
			
		}
	}

    hipfftDoubleComplex *data;
    hipMalloc((void**)&data, sizeof(hipfftDoubleComplex)*8*N);
    hipMemcpy(data, dp, sizeof(double)*8*N*2, hipMemcpyHostToDevice);
	
	//cufft
	if (hipfftPlan3d(&p1,NNx,NNy,NNz, HIPFFT_Z2Z) != HIPFFT_SUCCESS) {
        printf("CUFFT error: Plan creation failed.\n");
		exit(1);
    }
    if (hipfftExecZ2Z(p1, data, data, HIPFFT_FORWARD) != HIPFFT_SUCCESS) {
		printf("CUFFT error: ExecZ2Z forward failed.\n");
		exit(1);
    }
    hipMemcpy(fftdp, data, sizeof(double)*8*N*2, hipMemcpyDeviceToHost);
    hipDeviceSynchronize();
    hipfftDestroy(p1);
    hipFree(data);
	

	for(int i=0;i<NNx;i++){
		for(int j=0;j<NNy;j++){
			for(int k=0;k<NNz;k++){
				//Multiply 2 imaginary numbers
				int index = i*NNx*NNy + j*NNx + k;
				// phi[index][0] = fftdp[index][0] * fftgf[index][0] - fftdp[index][1] * fftgf[index][1];
				// phi[index][1] = fftdp[index][0] * fftgf[index][1] + fftdp[index][1] * fftgf[index][0];
				phi[index] = fftdp[index] * fftgf[index];
			}
		}
	}

	hipfftDoubleComplex *data2;
    hipMalloc((void**)&data2, sizeof(hipfftDoubleComplex)*8*N);
    hipMemcpy(data, phi, sizeof(double)*8*N*2, hipMemcpyHostToDevice);
	
	////ifft cufft
	if (hipfftPlan3d(&p2,NNx,NNy,NNz, HIPFFT_Z2Z) != HIPFFT_SUCCESS) {
        printf("CUFFT error: Plan creation failed.\n");
		exit(1);
    }
    if (hipfftExecZ2Z(p2, data2, data2, HIPFFT_BACKWARD) != HIPFFT_SUCCESS) {
		printf("CUFFT error: ExecZ2Z forward failed.\n");
		exit(1);
    }
    hipMemcpy(ifftphi, data2, sizeof(double)*8*N*2, hipMemcpyDeviceToHost);
    hipDeviceSynchronize();
    hipfftDestroy(p2);
    hipFree(data2);

	
	

	for(int i=0;i<Nx;i++){
		for(int j=0;j<Ny;j++){
			for(int k=0;k<Nz;k++){
				int index1 = i*Nx*Ny+j*Nx+k;//index for N size grid.
				int index2 = i*NNx*NNy + j*NNx + k;// index for 2N size grid.
				grid->phi[index1] = -1.0/grid->dx / (8*N) * abs(ifftphi[index2]);
			}
		}		
	}

	// for (int i=0; i < Nx; i++){
	// 	for (int j=0; j < Ny; j++){  
	// 		for(int k=0 ; k < Nz ; k++){
	// 			_2nd_order_diff_3d(grid, i, j,k);
	// 		}   
	// 	}
	// }

	//Use GPU to calculate force from potential.
	_2nd_order_diff_3d_cuda(grid);

	free(densityPad);
	free(dp);
	free(fftdp);
	free(phi);
	free(ifftphi);

}

void Weight(struct grid3D *grid,struct particle3D *particle,int type){

	//Initialize Density field
	for(int i=0 ; i < grid->N ; i++){
		grid->density[i]=0.0;
	}

	for(int i=0;i < particle->number;i++){
		int lx,ly,lz,sx,sy,sz;
		double shift = -grid->L/2;	//make (0,0) to be in the center of grid.
		lx = (particle->x[i]-shift)/grid->dx;
		ly = (particle->y[i]-shift)/grid->dy;
		lz = (particle->z[i]-shift)/grid->dz;

		if(type == 0){
			//NGP
			sx = particle->x[i]-shift-lx * grid->dx - 0.5*grid->dx + 1;
			sy = particle->y[i]-shift-ly * grid->dy - 0.5*grid->dy + 1;
			sz = particle->z[i]-shift-lz * grid->dz - 0.5*grid->dz + 1;
			grid->density[(lx+sx)*grid->Nx*grid->Ny + (ly+sy)*grid->Nx + (lz+sz) ] += particle->mass[i];
		}else if(type == 1){
			//CIC
			for(int zz=0;zz<2;zz++){
				for(int j=0;j<4;j++){
					int p = j / 2;
					int q = j % 2;
					double wFactor = (1-fabs(particle->x[i]-shift-(lx+p)*grid->dx)/grid->dx)*(1-fabs(particle->y[i]-shift-(ly+q)*grid->dy)/grid->dy);
					wFactor *= (1-fabs(particle->z[i]-shift-(lz+zz)*grid->dz)/grid->dz);
					grid->density[(lx+p)*grid->Nx*grid->Ny + (ly+q)*grid->Nx + (lz+zz) ] += particle->mass[i] * wFactor;
				}
			}
			
		}else if(type == 2){
			//TSC
			//xxx
			//xox
			//xxx
			lx = (particle->x[i]-shift+0.5*grid->dx)/grid->dx;	//Find the nearest point in lattice index.
			ly = (particle->y[i]-shift+0.5*grid->dy)/grid->dy;
			lz = (particle->z[i]-shift+0.5*grid->dz)/grid->dz;
			double weightX[3];	//Weight factor in x direction for 3 affected points
			double weightY[3];
			double weightZ[3];
			//Construct weighting factor
			for(int xx=-1;xx<2;xx++){
				double ddx = fabs(particle->x[i]-shift-(lx+xx)*grid->dx);
				if(ddx <= grid->dx/2){
					weightX[xx+1] = 3.0/4 - pow(ddx / grid->dx,2);
				}else if(ddx<= grid->dx/2*3.0){
					weightX[xx+1] = 0.5*pow(1.5-ddx / grid->dx,2);
				}else{
					printf("Should not be here");
					weightX[xx+1]=0.0;
				}
				
			}
			for(int yy=-1;yy<2;yy++){
				double ddy = fabs(particle->y[i]-shift-(ly+yy)*grid->dy);
				if(ddy <= grid->dy/2){
					weightY[yy+1] = 3.0/4 - pow(ddy / grid->dy,2);
				}else if(ddy <= grid->dy/2*3.0){
					weightY[yy+1] = 0.5*pow(1.5-ddy / grid->dy,2);
				}else{
					weightY[yy+1]=0.0;
				}
			}
			for(int zz=-1;zz<2;zz++){
				double ddz = fabs(particle->z[i]-shift-(lz+zz)*grid->dz);
				if(ddz <= grid->dz/2){
					weightZ[zz+1] = 3.0/4 - pow(ddz / grid->dz,2);
				}else if(ddz <= grid->dz/2*3.0){
					weightZ[zz+1] = 0.5*pow(1.5-ddz / grid->dz,2);
				}else{
					weightZ[zz+1]=0.0;
				}
			}
			//Weight mass into density
			int indx,indy,indz;
			for(int xx=-1;xx<2;xx++){
				for(int yy=-1;yy<2;yy++){
					for(int zz=-1;zz<2;zz++){
						//Account for periodic boundary
						indx = ((lx+xx)+grid->Nx)%grid->Nx;
						indy = ((ly+yy)+grid->Ny)%grid->Ny;
						indz = ((lz+zz)+grid->Nz)%grid->Nz;
						int index = indx*grid->Nx*grid->Ny + indy*grid->Nx + indz;
						grid->density[index]+=weightX[xx+1]*weightY[yy+1]*weightZ[zz+1]*particle->mass[i];
					}
					
				}
			}
		}
		
	}
}
void WeightForce(struct grid3D *grid,struct particle3D *particle,int type){
	//type = 0/1/2  => NGP/CIC/TSC
	for(int i=0;i < particle->number;i++){
		int lx,ly,lz,sx,sy,sz;
		double shift = -grid->L/2;	//make (0,0) to be in the center of grid.
		lx = (particle->x[i]-shift)/grid->dx;
		ly = (particle->y[i]-shift)/grid->dy;
		lz = (particle->z[i]-shift)/grid->dz;
		particle->Fx[i]=0.0;
		particle->Fy[i]=0.0;
		particle->Fz[i]=0.0;

		if(type == 0){
			sx = particle->x[i]-shift-lx * grid->dx - 0.5*grid->dx + 1;
			sy = particle->y[i]-shift-ly * grid->dy - 0.5*grid->dy + 1;
			sz = particle->z[i]-shift-lz * grid->dz - 0.5*grid->dz + 1;
			int pos = (lx+sx)*grid->Nx*grid->Ny + (ly+sy)*grid->Nx + (lz+sz) ;
			particle->Fx[i]=grid->Fx[pos]*particle->mass[i];
			particle->Fy[i]=grid->Fy[pos]*particle->mass[i];
			particle->Fz[i]=grid->Fz[pos]*particle->mass[i];
		}else if(type == 1){
			for(int zz=0;zz<2;zz++){
				for(int j=0;j<4;j++){
					int p = j / 2;
					int q = j % 2;
					double wFactor = (1-fabs(particle->x[i]-shift-(lx+p)*grid->dx)/grid->dx)*(1-fabs(particle->y[i]-shift-(ly+q)*grid->dy)/grid->dy);
					wFactor *= (1-fabs(particle->z[i]-shift-(lz+zz)*grid->dz)/grid->dz);
					int pos = (lx+p)*grid->Nx*grid->Ny + (ly+q)*grid->Nx + (lz+zz) ;
					particle->Fx[i] += grid->Fx[pos] * wFactor*particle->mass[i];
					particle->Fy[i] += grid->Fy[pos] * wFactor*particle->mass[i];
					particle->Fz[i] += grid->Fz[pos] * wFactor*particle->mass[i];
				}
			}
			
		}else if(type == 2){
			//TSC
			//xxx
			//xox
			//xxx
			lx = (particle->x[i]-shift+0.5*grid->dx)/grid->dx;	//Find the nearest point in lattice index.
			ly = (particle->y[i]-shift+0.5*grid->dy)/grid->dy;
			lz = (particle->z[i]-shift+0.5*grid->dz)/grid->dz;
			double weightX[3];	//Weight factor in x direction for 3 affected points
			double weightY[3];
			double weightZ[3];
			//Construct weighting factor
			for(int xx=-1;xx<2;xx++){
				double ddx = fabs(particle->x[i]-shift-(lx+xx)*grid->dx);
				if(ddx <= grid->dx/2){
					weightX[xx+1] = 3.0/4 - pow(ddx / grid->dx,2);
				}else if(ddx<= grid->dx/2*3.0){
					weightX[xx+1] = 0.5*pow(1.5-ddx / grid->dx,2);
				}else{
					printf("Should not be here");
					weightX[xx+1]=0.0;
				}
				
			}
			for(int yy=-1;yy<2;yy++){
				double ddy = fabs(particle->y[i]-shift-(ly+yy)*grid->dy);
				if(ddy <= grid->dy/2){
					weightY[yy+1] = 3.0/4 - pow(ddy / grid->dy,2);
				}else if(ddy <= grid->dy/2*3.0){
					weightY[yy+1] = 0.5*pow(1.5-ddy / grid->dy,2);
				}else{
					weightY[yy+1]=0.0;
				}
			}
			for(int zz=-1;zz<2;zz++){
				double ddz = fabs(particle->z[i]-shift-(lz+zz)*grid->dz);
				if(ddz <= grid->dz/2){
					weightZ[zz+1] = 3.0/4 - pow(ddz / grid->dz,2);
				}else if(ddz <= grid->dz/2*3.0){
					weightZ[zz+1] = 0.5*pow(1.5-ddz / grid->dz,2);
				}else{
					weightZ[zz+1]=0.0;
				}
			}
			//Weight mass into density
			int indx,indy,indz;
			for(int xx=-1;xx<2;xx++){
				for(int yy=-1;yy<2;yy++){
					for(int zz=-1;zz<2;zz++){
						//Account for periodic boundary
						indx = ((lx+xx)+grid->Nx)%grid->Nx;
						indy = ((ly+yy)+grid->Ny)%grid->Ny;
						indz = ((lz+zz)+grid->Nz)%grid->Nz;
						int index = indx*grid->Nx*grid->Ny + indy*grid->Nx + indz;
						double weight = weightX[xx+1]*weightY[yy+1]*weightZ[zz+1];
						particle->Fx[i] += grid->Fx[index]*weight*particle->mass[i];
						particle->Fy[i] += grid->Fy[index]*weight*particle->mass[i];
						particle->Fz[i] += grid->Fz[index]*weight*particle->mass[i];
					}
					
				}
			}
		}

		
	}
}
void kick(struct particle3D *particle , double dt){
	double ax,ay,az;
	for(int i=0 ; i<particle->number ; i++){
		//Cauculate the acceleration of each particle.
		ax = particle->Fx[i] / particle->mass[i];
		ay = particle->Fy[i] / particle->mass[i];
		az = particle->Fz[i] / particle->mass[i];

		//Calculate velocity of each particle.
		particle->vx[i] += ax * dt;
		particle->vy[i] += ay * dt;
		particle->vz[i] += az * dt;
	}
}
void drift(struct particle3D *particle , double dt){
	for(int i=0 ; i<particle->number ; i++){
		particle->x[i] += particle->vx[i] * dt;
		particle->y[i] += particle->vy[i] * dt;
		particle->z[i] += particle->vz[i] * dt;
	}
}
void init_rk4(struct particle3D *particle, struct particle3D *buff, struct rk43D *rk4){
	for(int i=0 ; i<particle->number ; i++){
		buff->vx[i] = particle->vx[i];
		buff->vy[i] = particle->vy[i];
		buff->vz[i] = particle->vz[i];
		buff->Fx[i] = particle->Fx[i];
		buff->Fy[i] = particle->Fy[i];
		buff->Fz[i] = particle->Fz[i];
		rk4->ax[i] = 0;
		rk4->ay[i] = 0;
		rk4->az[i] = 0;
		rk4->vx[i] = 0;
		rk4->vy[i] = 0;
		rk4->vz[i] = 0;
		rk4->step1 = 0;
		rk4->step2 = 1;
	}
}
void rk4_mid(struct particle3D *particle, struct particle3D *buff, struct rk43D *rk4, double dt, int weighting){
	for(int i=0 ; i<particle->number ; i++){
		buff->Fx[i] = rk4->step1 * buff->Fx[i] + rk4->step2 * particle->Fx[i];
		buff->Fy[i] = rk4->step1 * buff->Fy[i] + rk4->step2 * particle->Fy[i];
		buff->Fz[i] = rk4->step1 * buff->Fz[i] + rk4->step2 * particle->Fz[i];

		buff->x[i] = particle->x[i] + buff->vx[i] * dt;
		buff->y[i] = particle->y[i] + buff->vy[i] * dt;
		buff->z[i] = particle->z[i] + buff->vz[i] * dt;

		rk4->ax[i] += weighting/6.0 * buff->Fx[i]/particle->mass[i];
		rk4->ay[i] += weighting/6.0 * buff->Fy[i]/particle->mass[i];
		rk4->az[i] += weighting/6.0 * buff->Fz[i]/particle->mass[i];
		rk4->vx[i] += weighting/6.0 * buff->vx[i];
		rk4->vy[i] += weighting/6.0 * buff->vy[i];
		rk4->vz[i] += weighting/6.0 * buff->vz[i];

		buff->vx[i] = particle->vx[i] + buff->Fx[i]/particle->mass[i] * dt;
		buff->vy[i] = particle->vy[i] + buff->Fy[i]/particle->mass[i] * dt;
		buff->vz[i] = particle->vz[i] + buff->Fz[i]/particle->mass[i] * dt;

		rk4->step1 = 1;
		rk4->step2 = 0;
	}
}
void rk4_end(struct particle3D *particle, struct rk43D *rk4, double dt){
	for(int i=0 ; i<particle->number ; i++){
		particle->x[i]  += rk4->vx[i] * dt;
		particle->y[i]  += rk4->vy[i] * dt;
		particle->z[i]  += rk4->vz[i] * dt;
		particle->vx[i] += rk4->ax[i] * dt;
		particle->vy[i] += rk4->ay[i] * dt;
		particle->vz[i] += rk4->az[i] * dt;
	}
}
void periodic_boundary(double position, double length){
	int sign = position/abs(position);
	position = sign * remainder(abs(position + sign*length/2), length) - sign*length/2;
	cout << "A particle reaches the boundary." << endl;
}
void boundary_check(int boundary, struct particle3D *particle, double L){
	if (boundary == 0){
		for (int i=0; i<particle->number; i++){
			if ( abs(particle->x[i]) > L/2){
				periodic_boundary(particle->x[i],L);
			}
			if ( abs(particle->y[i]) > L/2){
				periodic_boundary(particle->y[i],L);
			}
			if ( abs(particle->z[i]) > L/2){
				periodic_boundary(particle->z[i],L);
			}
		}
	}
	else if (boundary == 1){
		for (int i=0; i<particle->number; i++){
			if ( abs(particle->x[i]) > L/2 || abs(particle->y[i]) > L/2 || abs(particle->z[i]) > L/2){
				particle->mass[i] = 0;
				cout << "A particle reaches the boundary." << endl;
			}
		}
	}
}
//Functions to locate memory and free memory of different struct.
void locateMemoryParticle(struct particle3D *particle,int N){
	particle->mass = (double*)malloc(N*sizeof(double));
    particle->x = (double*)malloc(N*sizeof(double));
    particle->y = (double*)malloc(N*sizeof(double));
    particle->z = (double*)malloc(N*sizeof(double));
	particle->Fx = (double*)malloc(N*sizeof(double));
    particle->Fy = (double*)malloc(N*sizeof(double));
    particle->Fz = (double*)malloc(N*sizeof(double));
	particle->vx = (double*)malloc(N*sizeof(double));
    particle->vy = (double*)malloc(N*sizeof(double));
	particle->vz = (double*)malloc(N*sizeof(double));
}
void freeMemoryParticle(struct particle3D *particle){
	free(particle->mass);
	free(particle->x);
	free(particle->y);
	free(particle->z);
	free(particle->Fx);
	free(particle->Fy);
	free(particle->Fz);
	free(particle->vx);
	free(particle->vy);
	free(particle->vz);
}
void locateMemoryRk4(struct rk43D *rk4,int N){
	rk4->ax = (double*)malloc(N*sizeof(double));
	rk4->ay = (double*)malloc(N*sizeof(double));
	rk4->az = (double*)malloc(N*sizeof(double));
	rk4->vx = (double*)malloc(N*sizeof(double));
	rk4->vy = (double*)malloc(N*sizeof(double));
	rk4->vz = (double*)malloc(N*sizeof(double));	
}
void freeMemoryRk4(struct rk43D *rk4){
	free(rk4->ax);
	free(rk4->ay);
	free(rk4->az);
	free(rk4->vx);
	free(rk4->vy);
	free(rk4->vz);
}
void locateMemoryGrid(struct grid3D *grid){
	grid->density = (double*) malloc(grid->N*sizeof(double));	//Density on the grid
	grid->phi = (double*) malloc(grid->N*sizeof(double));		//Potential on the grid
	grid->Fx = (double*) malloc(grid->N*sizeof(double));			//Force on grid comes from potential calculation
	grid->Fy = (double*) malloc(grid->N*sizeof(double));
	grid->Fz = (double*) malloc(grid->N*sizeof(double));
}
void freeMemoryGrid(struct grid3D *grid){
	free(grid->density);
	free(grid->phi);
	free(grid->Fx);
	free(grid->Fy);
	free(grid->Fz);
}
